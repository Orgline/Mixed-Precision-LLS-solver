#include "hip/hip_runtime.h"
#include <iostream>

__global__
void seteye( int m, int n, float *a, int lda)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m && j < n) {
		if (i == j) 
			a[i+j*lda] = 1;
		else
			a[i+j*lda] = 0;
	}

}

void qrSolve(int m, int n, float *A, int lda, float *R, int ldr, float *reflector, float *d_tau,float *d_work, int lwork)
{
    printf("Function qrSolve\n");
    hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    

	int *devInfo = 0;
    hipMalloc ((void**)&devInfo, sizeof(int));
    cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    
    //cout << "lwork/4 (elms)=" << lwork/4 << endl;
    
    //hipEvent_t start, stop;
    startTimer();

    cusolver_status = hipsolverDnSgeqrf(
        cusolverH, 
        m, 
        n, 
        A, 
        lda, 
        d_tau, 
        d_work, 
        lwork, 
        devInfo);

    printf("Sgeqrf return code %d\n", cusolver_status);

    float milliseconds =  stopTimer();

    int info;
    hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    printf("cuSolver: sgeqrf (%d,%d) takes %f ms\n",m,n,milliseconds);
    
    printf("TFLOPS/s: %f\n",(2.0*m*n*n - 2.0/3*n*n*n)/milliseconds*1000/1e12);

    printf("INFO %d\n",info);

    copyDevice(m,n,A,lda,R,ldr);
    clearTri(n,n,R,ldr);
    //printMatrixDevice("RR.csv",n,n,R,n);

    float *tA;
    hipMalloc(&tA,sizeof(float)*m*n);
    hipMemcpy(tA,A,sizeof(float)*m*n,hipMemcpyDeviceToDevice);
    hipMemcpy(reflector,A,sizeof(float)*m*n,hipMemcpyDeviceToDevice);
    dim3 grid222( (m+1)/32, (n+1)/32 );
    dim3 block222( 32, 32 );
    seteye<<<grid222,block222>>>( m, n, tA, m);
    //printMatrixDevice("tA.csv",m,n,tA,m);

    startTimer();
    cusolver_status= hipsolverDnSormqr(
        cusolverH,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_OP_N,
        m,
        n,
        n,
        A,
        lda,
        d_tau,
        tA,
        m,
        d_work,
        lwork,
        devInfo);
    milliseconds = stopTimer();
    printf("Obtaining Q takes %f ms\n",milliseconds);
    hipMemcpy(A,tA,sizeof(float)*m*n,hipMemcpyDeviceToDevice);
    hipFree(tA);
}

void sllsSolve(int m,int n,float* reflector, int lda, float* dtau,float* d_work,int lwork, float *R, float *dfx)
{
    printf("1\n");
    int *devInfo = 0;
    hipMalloc ((void**)&devInfo, sizeof(int));

    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnCreate(&cusolverH);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    

    float hfb[m];
    for(int i=0;i<m;i++)
    {
        hfb[i] = 1.0;
    }
    float *dfb;
    hipMalloc(&dfb,sizeof(float)*m);
    
    hipMemcpy(dfb,hfb,sizeof(float)*m,hipMemcpyHostToDevice);
    //free(hfb);
    printf("2\n");

    startTimer();
    hipsolverDnSormqr(
        cusolverH,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_OP_T,
        m,
        1,
        n,
        reflector,
        lda,
        dtau,
        dfb,
        m,
        d_work,
        lwork,
        devInfo);

    dim3 grids2d2( (n+31)/32, (1+31)/32 );
    dim3 blocks2d2( 32, 32 );
    myslacpy<<<grids2d2,blocks2d2>>>(n, 1, dfb, m, dfx, n );
    float sone = 1.0;
    hipblasStrsm(handle,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,n,1,&sone,R,n,dfx,n);
    float milliseconds;
    milliseconds = stopTimer();
    printf("single LLS direct solver takes %fms\n",milliseconds);
    hipFree(dfb);
}

void checkResult(int m, int n, float *A, int lda,float *x)
{
    startTimer();
    double *db,*dx;
    double hb[m];
    for(int i=0;i<m;i++)
    {
        hb[i] = 1.0;
    }
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double normx;
    hipMalloc(&db,sizeof(double)*m);
    hipMalloc(&dx,sizeof(double)*n);
    hipMemcpy(db,hb,m*sizeof(double),hipMemcpyHostToDevice);

    double done = 1.0;
    double dnegone = -1.0;
    double dzero = 0.0;
    
    double *doubleA;
    hipMalloc(&doubleA,sizeof(double)*m*n);

    dim3 grids2d( (m+31)/32, (n+31)/32 );
    dim3 blocks2d( 32, 32 );
    s2d<<<grids2d,blocks2d>>>( m, n, A,m,doubleA,m);
    hipFree(A);

    double *doubleX;
    hipMalloc(&doubleX,sizeof(double)*n);
    float milliseconds  = stopTimer();
    printf("Phase 1 takes %fms\n",milliseconds);
    dim3 grids2d2( (n+31)/32, (1+31)/32 );
    dim3 blocks2d2( 32, 32 );
    s2d<<<grids2d2,blocks2d2>>>( n, 1, x,n,doubleX,n);
    hipFree(x);

    startTimer();
    double *tempx;
    hipMalloc(&tempx,sizeof(double)*n);
    hipblasDgemv(handle,HIPBLAS_OP_T,m,n,&done,doubleA,m,db,1,&dzero,tempx,1);
    double normb;
    normb = dnorm(m,1,tempx);
    hipFree(tempx);

    hipblasDgemv(handle,HIPBLAS_OP_N,m,n,&done,doubleA,m,doubleX,1,&dnegone,db,1);
    hipblasDgemv(handle,HIPBLAS_OP_T,m,n,&done,doubleA,m,db,1,&dzero,doubleX,1);
    double normsol = dnorm(n,1,doubleX);
    printf("||A^T*(Ax-b)||/(||A^T*b||) = %.6e\n",normsol/normb);
    milliseconds = stopTimer();
    printf("Phase 2 takes %fms\n",milliseconds);
    
}


void sSolveLLS(int m,int n, float* hA,int lda)
{
    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnCreate(&cusolverH);

    printf("Function sSolveLLS\n");
    float *dA;
    hipMalloc(&dA,sizeof(float)*n*m);
    host2Device(m,n,dA,hA);
    float *R;
    hipMalloc(&R,sizeof(float)*n*n);

    float *reflector;
    hipMalloc(&reflector,sizeof(float)*m*n);

    float *d_tau;
    hipMalloc(&d_tau, sizeof(float)*n);
    
    float *d_work;
    int lwork = 0;
    
    hipsolverDnSgeqrf_bufferSize(
		cusolverH, 
		m, 
		n, 
		dA, 
		lda, 
        &lwork);
    hipMalloc((void**)&d_work, sizeof(float)*lwork);

    qrSolve(m,n,dA,m,R,n,reflector,d_tau,d_work,lwork);
    //printMatrixDevice("Q.csv",m,n,dA,m);
    //printMatrixDevice("R.csv",n,n,R,n);
    float *A;
    hipMalloc(&A,sizeof(float)*n*m);
    host2Device(m,n,A,hA);
    //printMatrixDevice("A.csv",m,n,A,m);

    float *dfx;
    hipMalloc(&dfx,sizeof(float)*n);

    //check QR result
    float normA = snorm(m,n,A);
    sgemm(m,n,n,dA,m,R,n,A,m,1.0,-1.0);
    float normResult = snorm(m,n,A);
    printf("||A-QR||/||A|| is %.3e\n", normResult/normA);
    hipFree(dA);

    sllsSolve(m,n,reflector, lda, d_tau, d_work, lwork, R, dfx);
    host2Device(m,n,A,hA);
    //printMatrixDevice("A.csv",m, n, A, m);
    //printMatrixDevice("x.csv",n, 1, dfx, n);
    checkResult(m, n, A, lda,dfx);
   
}